#include "hip/hip_runtime.h"
#include "Functions.h"
#include <iostream>
//Profiling
#include <time.h>

//Fixt intellisense problemen
#include <hip/hip_runtime.h>
#include ""

__global__ void unwrap_gpu(float* d_Output, float* d_image, int image_edge,float cutoff){
	//Elke thread krijgt 1 kolom
		int column=(threadIdx.x)+blockDim.x*(blockIdx.x);
		if(column < image_edge){
			float deltaref=0;
			d_Output[column*image_edge]=d_image[column*image_edge];
			for(int row=0;row<image_edge-1;row++){
				float delta=d_image[column*image_edge+row+1]-d_image[column*image_edge+row];
				if(fabsf(delta)>=cutoff)
					deltaref=deltaref-delta;
				d_Output[column*image_edge+row+1]=d_image[column*image_edge+row+1]+deltaref;
			}
		}
		//Other direction
		if(column <= image_edge){
			float deltaref=0;
			for(int row=0;row<=image_edge;row++){
				float delta=d_Output[column+row*image_edge+image_edge]-d_Output[column+row*image_edge];
				if(fabsf(delta)>=cutoff)
					deltaref=deltaref-delta;
				d_Output[column+row*image_edge]=d_Output[column+row*image_edge+image_edge]+deltaref;
			}
		}
}

int main ()
{
	//Useful constants
	int image_size=768;
	int elements_amount=(int) pow((float)image_size,2);
	int sample_size=100;


	//Read the binary file containing the wrapped & flattened image
	float* Data;
	Data=read_data("C:\\Users\\janmorez\\Dropbox\\MATLAB\\CUDA\\Image_Wrapped.bin");

	//=============CPU-unwrap=============
	clock_t begin,end;

	if(image_size < 1024){
		float* CPUTimeArray=(float*)malloc(sizeof(float)*sample_size);
		float * OutputCPU;
		for(int i=0;i<sample_size;i++){
			begin=clock();
			OutputCPU=unwrap_cpu(Data,image_size);
			end=clock();
			float CPUTime=1000*(float)(end-begin)/CLOCKS_PER_SEC;
			CPUTimeArray[i]=CPUTime;
			printf("CPU Unwrap time was: %f ms \n",CPUTime);
		}
		export_data("C:\\Users\\janmorez\\Dropbox\\MATLAB\\CUDA\\OutputCPU.bin",OutputCPU,elements_amount);
		export_data("C:\\Users\\janmorez\\Dropbox\\MATLAB\\CUDA\\OutputCPUTimes.bin",CPUTimeArray,sample_size);

		free(OutputCPU);
		free(CPUTimeArray);
		}


	//=============GPU-unwrap=============
	//Determine gridsize & dimensions
	printf("\n =============================\n Starting GPU-Unwrap \n");
	hipDeviceProp_t device;
	hipGetDeviceProperties(&device,0);
	int MaxThreadsPerBlock=device.maxThreadsPerBlock;
	dim3 BlockDim;
	BlockDim.x=MaxThreadsPerBlock;
	int GridSize=ceil((float)image_size/MaxThreadsPerBlock);
	printf("Amount of threads per block: %i. Gridsize (in blocks): %i \n",MaxThreadsPerBlock,GridSize);

	//Profiling code
	hipEvent_t startG,stopG;
	hipEventCreate(&startG);
	hipEventCreate(&stopG);
	float* GPUTimeArray=(float*)malloc((size_t) sizeof(float)*sample_size);
	float timeG;
	float* h_OutputGPU;

	for(int i=0;i< sample_size;i++){
	//Allocate device Output memory
	hipError_t err=hipSuccess;
	float* d_Output=NULL;
	err=hipMalloc(&d_Output,elements_amount*sizeof(float));
	if(err==hipSuccess){
		printf("Starting GPU-unwrap... \n");
		hipEventRecord(startG,0);
		float* d_Data=copy_data_to_device(Data,image_size);
		unwrap_gpu<<<GridSize,BlockDim>>>(d_Output,d_Data,image_size,1);
		hipDeviceSynchronize();
		hipDeviceSynchronize();
		h_OutputGPU=copy_data_to_host(d_Output,image_size);	
		hipEventRecord(stopG);
		hipFree(d_Output);
		hipFree(d_Data);
		printf("GPU-unwrap completed. Possible kernel errors: %s. \n",hipGetErrorString(hipGetLastError()));

	}
	else
		printf("GPU memory allocation failed: %s \n",hipGetErrorString(err));


	hipEventSynchronize(stopG);
	hipEventElapsedTime(&timeG,startG,stopG);
	GPUTimeArray[i]=timeG;
	printf("Complete GPU-unwrap took: %f ms \n",GPUTimeArray[i]);
	}
	

	export_data("C:\\Users\\janmorez\\Dropbox\\MATLAB\\CUDA\\OutputGPU.bin",h_OutputGPU,elements_amount);
	export_data("C:\\Users\\janmorez\\Dropbox\\MATLAB\\CUDA\\OutputGPUTimes.bin",GPUTimeArray,sample_size);

	free(Data);
	free(GPUTimeArray);
	//=============END OF PROGRAM=========
	printf("Program ended. Press any key to close this window... \n ");
	getchar(); 
	return 0;
	
}